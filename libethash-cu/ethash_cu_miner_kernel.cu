#include "hip/hip_runtime.h"
/*
* Genoil's CUDA mining kernel for Ethereum
* based on Tim Hughes' opencl kernel.
* thanks to trpuvot,djm34,sp,cbuchner for things i took from ccminer.
*/

#define SHUFFLE_MIN_VER 350

#include "ethash_cu_miner_kernel.h"
#include "ethash_cu_miner_kernel_globals.h"
#include "rotl64.cuh"
#include "cuda_helper.h"
#include "keccak.cuh"
#include ""
#include "hip/device_functions.h"
#include "hip/hip_vector_types.h"

#define ACCESSES 64
#define THREADS_PER_HASH (128 / 16)

#define FNV_PRIME	0x01000193

#define SWAP64(v) \
  ((ROTL64L(v,  8) & 0x000000FF000000FF) | \
   (ROTL64L(v, 24) & 0x0000FF000000FF00) | \
   (ROTL64H(v, 40) & 0x00FF000000FF0000) | \
   (ROTL64H(v, 56) & 0xFF000000FF000000))

#define PACK64(result, lo, hi) asm("mov.b64 %0, {%1,%2};//pack64"  : "=l"(result) : "r"(lo), "r"(hi));
#define UNPACK64(lo, hi, input) asm("mov.b64 {%0, %1}, %2;//unpack64" : "=r"(lo),"=r"(hi) : "l"(input));

#define copy(dst, src, count) for (uint32_t i = 0; i < count; i++) { (dst)[i] = (src)[i]; }

#define countof(x) (sizeof(x) / sizeof(x[0]))

#define fnv(x,y) ((x) * FNV_PRIME ^(y))

__device__ uint4 fnv4(uint4 a, uint4 b)
{
	uint4 c;
	c.x = a.x * FNV_PRIME ^ b.x;	
	c.y = a.y * FNV_PRIME ^ b.y;
	c.z = a.z * FNV_PRIME ^ b.z;
	c.w = a.w * FNV_PRIME ^ b.w;
	return c;
}

__device__ uint32_t fnv_reduce(uint4 v)
{
	return fnv(fnv(fnv(v.x, v.y), v.z), v.w);
}


__device__ hash64_t init_hash(hash32_t const* header, uint64_t nonce)
{
	hash64_t init;

	// sha3_512(header .. nonce)
	uint64_t state[25];

	copy(state, header->uint64s, 4);
	state[4] = nonce;
	state[5] = 0x0000000000000001;
	state[6] = 0;
	state[7] = 0;
	state[8] = 0x8000000000000000;
	for (uint32_t i = 9; i < 25; i++)
	{
		state[i] = 0;
	}
	
	keccak_f1600_block((uint2 *)state, 8);
	copy(init.uint64s, state, 8);
	return init;
}

__device__ uint32_t inner_loop(uint4 mix, uint32_t thread_id, uint32_t* share, hash128_t const* g_dag)
{
	// share init0
	if (thread_id == 0)
		*share = mix.x;

	uint32_t init0 = *share;
	
	uint32_t a = 0;

	do
	{
		
		bool update_share = thread_id == ((a >> 2) & (THREADS_PER_HASH-1));

		//#pragma unroll 4
		for (uint32_t i = 0; i < 4; i++)
		{

			if (update_share)
			{
				uint32_t m[4] = { mix.x, mix.y, mix.z, mix.w };
				*share = fnv(init0 ^ (a + i), m[i]) % d_dag_size;
			}
			__threadfence_block();

#if __CUDA_ARCH__ >= 350
			mix = fnv4(mix, __ldg(&g_dag[*share].uint4s[thread_id]));
#else
			mix = fnv4(mix, g_dag[*share].uint4s[thread_id]);
#endif
			
		}
		
	} while ((a += 4) != ACCESSES);
	
	return fnv_reduce(mix);
}

__device__ hash32_t final_hash(hash64_t const* init, hash32_t const* mix)
{
	uint64_t state[25];

	hash32_t hash;

	// keccak_256(keccak_512(header..nonce) .. mix);
	copy(state, init->uint64s, 8);
	copy(state + 8, mix->uint64s, 4);
	state[12] = 0x0000000000000001;
	for (uint32_t i = 13; i < 16; i++)
	{
		state[i] = 0;
	}
	state[16] = 0x8000000000000000;
	for (uint32_t i = 17; i < 25; i++)
	{
		state[i] = 0;
	}

	keccak_f1600_block((uint2 *)state, 1);

	// copy out
	copy(hash.uint64s, state, 4);
	return hash;
}

typedef union
{
	hash64_t init;	
	hash32_t mix;
} compute_hash_share;

#if __CUDA_ARCH__ >= SHUFFLE_MIN_VER
__device__ uint64_t compute_hash_shuffle(
	hash32_t const* g_header,
	hash128_t const* g_dag,
	uint64_t nonce
	)
{
	// sha3_512(header .. nonce)
	uint64_t state[25];
	
	copy(state, g_header->uint64s, 4);
	state[4] = nonce;
	state[5] = 0x0000000000000001ULL;
	for (uint32_t i = 6; i < 25; i++)
	{
		state[i] = 0;
	}
	state[8] = 0x8000000000000000ULL;
	keccak_f1600_block((uint2 *)state, 8);

	// Threads work together in this phase in groups of 8.
	const int thread_id = threadIdx.x & (THREADS_PER_HASH - 1);
	const int start_lane = threadIdx.x & ~(THREADS_PER_HASH - 1);
	const int mix_idx = (thread_id & 3); 

	uint4 mix;

	uint32_t shuffle[16];
	//uint32_t * init = (uint32_t *)state;

	uint32_t init[16];
	UNPACK64(init[0], init[1], state[0]);
	UNPACK64(init[2], init[3], state[1]);
	UNPACK64(init[4], init[5], state[2]);
	UNPACK64(init[6], init[7], state[3]);
	UNPACK64(init[8], init[9], state[4]);
	UNPACK64(init[10], init[11], state[5]);
	UNPACK64(init[12], init[13], state[6]);
	UNPACK64(init[14], init[15], state[7]);

	for (int i = 0; i < THREADS_PER_HASH; i++)
	{

		// share init among threads
		for (int j = 0; j < 16; j++)
			shuffle[j] = __shfl(init[j], start_lane + i);
		
		// ugly but avoids local reads/writes
		if (mix_idx == 0) {
			mix = make_uint4(shuffle[0], shuffle[1], shuffle[2], shuffle[3]);			
		}
		else if (mix_idx == 1) {
			mix = make_uint4(shuffle[4], shuffle[5], shuffle[6], shuffle[7]);
		}
		else if (mix_idx == 2) {
			mix = make_uint4(shuffle[8], shuffle[9], shuffle[10], shuffle[11]);
		}
		else {
			mix = make_uint4(shuffle[12], shuffle[13], shuffle[14], shuffle[15]);
		}
		
		uint32_t init0 = __shfl(shuffle[0], start_lane);
		
		
		for (uint32_t a = 0; a < ACCESSES; a+=4)
		{
			int t = ((a >> 2) & (THREADS_PER_HASH - 1));

			for (uint32_t b = 0; b < 4; b++)
			{
				if (thread_id == t)
				{
					shuffle[0] = fnv(init0 ^ (a + b), ((uint32_t *)&mix)[b]) % d_dag_size;;
				}
				shuffle[0] = __shfl(shuffle[0], start_lane + t);

				mix = fnv4(mix, g_dag[shuffle[0]].uint4s[thread_id]);			
			}
		} 

		uint32_t thread_mix = fnv_reduce(mix);

		// update mix accross threads

		for (int j = 0; j < 8; j++)
			shuffle[j] = __shfl(thread_mix, start_lane + j);

		if (i == thread_id) {	

			//move mix into state:
			PACK64(state[8],  shuffle[0], shuffle[1]);
			PACK64(state[9],  shuffle[2], shuffle[3]);
			PACK64(state[10], shuffle[4], shuffle[5]);
			PACK64(state[11], shuffle[6], shuffle[7]);
		}
		
	}

	// keccak_256(keccak_512(header..nonce) .. mix);
	state[12] = 0x0000000000000001ULL;
	for (uint32_t i = 13; i < 25; i++)
	{
		state[i] = 0ULL;
	}
	state[16] = 0x8000000000000000;
	keccak_f1600_block((uint2 *)state, 1);

	return state[0];
}
#endif

__device__ hash32_t compute_hash(
	hash32_t const* g_header,
	hash128_t const* g_dag,
	uint64_t nonce
	)
{
	extern __shared__  compute_hash_share share[];

	// Compute one init hash per work item.
	hash64_t init = init_hash(g_header, nonce);

	// Threads work together in this phase in groups of 8.
	uint32_t const thread_id = threadIdx.x & (THREADS_PER_HASH-1);
	uint32_t const hash_id   = threadIdx.x >> 3;

	hash32_t mix;

	for (int i = 0; i < THREADS_PER_HASH; i++)
	{
		// share init with other threads
		if (i == thread_id)
			share[hash_id].init = init;
		
		uint4 thread_init = share[hash_id].init.uint4s[thread_id & 3];
		
		uint32_t thread_mix = inner_loop(thread_init, thread_id, share[hash_id].mix.uint32s, g_dag);

		share[hash_id].mix.uint32s[thread_id] = thread_mix;
		

		if (i == thread_id)
			mix = share[hash_id].mix;
	}

	return final_hash(&init, &mix);
}

__global__ void 
__launch_bounds__(128, 7)
ethash_search(
	uint32_t* g_output,
	hash32_t const* g_header,
	hash128_t const* g_dag,
	uint64_t start_nonce,
	uint64_t target
	)
{
	
	uint32_t const gid = blockIdx.x * blockDim.x + threadIdx.x;	
	
#if __CUDA_ARCH__ >= SHUFFLE_MIN_VER
	uint64_t hash = compute_hash_shuffle(g_header, g_dag, start_nonce + gid);
	if (cuda_swab64(hash) < target)
	{
		atomicInc(g_output, d_max_outputs);
		g_output[g_output[0]] = gid;
	}
#else
	hash32_t hash = compute_hash(g_header, g_dag, start_nonce + gid);	
	if (cuda_swab64(hash.uint64s[0]) < target)
	{
		atomicInc(g_output,d_max_outputs);
		g_output[g_output[0]] = gid;
	}
#endif
	
	
}

void run_ethash_hash(
	hash32_t* g_hashes,
	hash32_t const* g_header,
	hash128_t const* g_dag,
	uint64_t start_nonce
) 
{
}

void run_ethash_search(
	uint32_t blocks,
	uint32_t threads,
	hipStream_t stream,
	uint32_t* g_output,
	hash32_t const* g_header,
	hash128_t const* g_dag,
	uint64_t start_nonce,
	uint64_t target
)
{
#if __CUDA_ARCH__ >= SHUFFLE_MIN_VER
	ethash_search <<<blocks, threads, 0, stream >>>(g_output, g_header, g_dag, start_nonce, target);
#else
	ethash_search <<<blocks, threads, (sizeof(compute_hash_share) * threads) / THREADS_PER_HASH, stream>>>(g_output, g_header, g_dag, start_nonce, target);
#endif
}

hipError_t set_constants(
	uint32_t * dag_size,
	uint32_t * max_outputs
	)
{
	hipError_t result;
	result = hipMemcpyToSymbol(HIP_SYMBOL(d_dag_size), dag_size, sizeof(uint32_t));
	result = hipMemcpyToSymbol(HIP_SYMBOL(d_max_outputs), max_outputs, sizeof(uint32_t));
	return result;
}
