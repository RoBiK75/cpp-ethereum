#include "hip/hip_runtime.h"
/*
* Genoil's CUDA mining kernel for Ethereum
* based on Tim Hughes' opencl kernel.
* thanks to sp_, trpuvot, djm34, cbuchner for things i took from ccminer.
*/

#include "ethash_cu_miner_kernel.h"
#include "ethash_cu_miner_kernel_globals.h"
#include "cuda_helper.h"

#define SHUFFLE_MIN_VER 350
#if __CUDA_ARCH__ >= SHUFFLE_MIN_VER
#include "dagger_shuffled.cuh"
#else
#include "dagger_shared.cuh"
#endif

__global__ void 
__launch_bounds__(128, 7)
ethash_search(
	uint32_t* g_output,
	hash32_t const* g_header,
	hash128_t const* g_dag,
	uint64_t start_nonce,
	uint64_t target
	)
{
	
	uint32_t const gid = blockIdx.x * blockDim.x + threadIdx.x;	
	
#if __CUDA_ARCH__ >= SHUFFLE_MIN_VER
	uint64_t hash = compute_hash_shuffle(g_header, g_dag, start_nonce + gid);
	if (cuda_swab64(hash) < target)
#else
	hash32_t hash = compute_hash(g_header, g_dag, start_nonce + gid);	
	if (cuda_swab64(hash.uint64s[0]) < target)
#endif
	{
		atomicInc(g_output, d_max_outputs);
		g_output[g_output[0]] = gid;
	}
	
}

void run_ethash_search(
	uint32_t blocks,
	uint32_t threads,
	hipStream_t stream,
	uint32_t* g_output,
	hash32_t const* g_header,
	hash128_t const* g_dag,
	uint64_t start_nonce,
	uint64_t target
)
{
#if __CUDA_ARCH__ >= SHUFFLE_MIN_VER
	ethash_search <<<blocks, threads, 0, stream >>>(g_output, g_header, g_dag, start_nonce, target);
#else
	ethash_search <<<blocks, threads, (sizeof(compute_hash_share) * threads) / THREADS_PER_HASH, stream>>>(g_output, g_header, g_dag, start_nonce, target);
#endif
}

hipError_t set_constants(
	uint32_t * dag_size,
	uint32_t * max_outputs
	)
{
	hipError_t result;
	result = hipMemcpyToSymbol(HIP_SYMBOL(d_dag_size), dag_size, sizeof(uint32_t));
	result = hipMemcpyToSymbol(HIP_SYMBOL(d_max_outputs), max_outputs, sizeof(uint32_t));
	return result;
}
